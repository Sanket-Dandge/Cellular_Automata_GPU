#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "kernels.hpp"

using namespace std;

namespace kernels::gol {
    enum States : uint8_t {
        DEAD = 0,
        ALIVE = 1,
    };

    __device__ int count_neighbors(uint8_t *current_grid, uint col, uint row, int ca_grid_size) {
        uint left_col = (col - 1 + ca_grid_size) % ca_grid_size;
        uint right_col = (col + 1) % ca_grid_size;
        uint row_offset = row * ca_grid_size;
        uint top_row_offset = ((row - 1 + ca_grid_size) % ca_grid_size) * ca_grid_size;
        uint bottom_row_offset = ((row + 1) % ca_grid_size) * ca_grid_size;

        return current_grid[left_col + top_row_offset] + current_grid[col + top_row_offset] +
               current_grid[right_col + top_row_offset] +
               current_grid[left_col + bottom_row_offset] + current_grid[col + bottom_row_offset] +
               current_grid[right_col + bottom_row_offset] + current_grid[left_col + row_offset] +
               current_grid[right_col + row_offset];
    }

    __global__ void compute_next_gen_kernel(uint8_t *current_grid, uint8_t *next_grid,
                                            int ca_grid_size) {
        uint col = blockIdx.x * blockDim.x + threadIdx.x;
        uint row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t row_offset = row * ca_grid_size;
        uint index = row_offset + col;
        if (index >= ca_grid_size * ca_grid_size) {
            printf("%d,%d\n", col, row);
        }
        int living_neighbors = kernels::gol::count_neighbors(current_grid, col, row, ca_grid_size);
        next_grid[index] =
            living_neighbors == 3 || (living_neighbors == 2 && current_grid[index]) ? ALIVE : DEAD;
        return;
    }

    void compute_next_gen(uint8_t *current_grid, uint8_t *next_grid, size_t ca_grid_size,
                          size_t niter) {
        // Allocate device memory
        uint8_t *d_current = nullptr, *d_next = nullptr;
        size_t total_size = ca_grid_size * ca_grid_size;
        CUDA_CHECK(hipMalloc(&d_current, total_size * sizeof(bool)));
        CUDA_CHECK(hipMalloc(&d_next, total_size * sizeof(bool)));

        // Copy data to device
        CUDA_CHECK(
            hipMemcpy(d_current, current_grid, total_size * sizeof(bool), hipMemcpyHostToDevice));

        // Launch kernel
        dim3 block_size(32, 32);
        dim3 grid_size((ca_grid_size + block_size.x - 1) / block_size.x,
                       (ca_grid_size + block_size.y - 1) / block_size.y);

        int citers = 0;
        while (citers < niter) {
            kernels::gol::compute_next_gen_kernel<<<grid_size, block_size>>>(d_current, d_next,
                                                                             ca_grid_size);
            hipDeviceSynchronize();
            niter++;
            swap(d_current, d_next);
        }

        CUDA_CHECK(hipGetLastError());
        hipDeviceSynchronize();

        // Copy result back to host
        CUDA_CHECK(
            hipMemcpy(next_grid, d_next, total_size * sizeof(bool), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_current));
        CUDA_CHECK(hipFree(d_next));
    }

} // namespace kernels::gol
