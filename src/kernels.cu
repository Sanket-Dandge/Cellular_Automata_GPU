#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "kernels.hpp"
#include "cyclic_ca.hpp"
#include "utils.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>

#define ROW_SIZE GRID_SIZE/ELEMENTS_PER_CELL    // Real grid dimension

#define CUDA_CHECK(call)                                                                           \
    {                                                                                              \
        hipError_t err = (call);                                                                  \
        if (err != hipSuccess) {                                                                  \
            printf("CUDA Error: %s at %s: %d\n", hipGetErrorString(err), __FILE__, __LINE__);     \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    }

namespace kernels {
    __host__ __device__ int count_neighbors(bool *current_grid, int col, int row, int grid_size) {
        int left_col = (col - 1 + grid_size) % grid_size;
        int right_col = (col + 1) % grid_size;
        int row_offset = row * grid_size;
        int top_row_offset = ((row - 1 + grid_size) % grid_size) * grid_size;
        int bottom_row_offset = ((row + 1) % grid_size) * grid_size;

        return current_grid[left_col + top_row_offset] + current_grid[col + top_row_offset] +
               current_grid[right_col + top_row_offset] + current_grid[left_col + bottom_row_offset] +
               current_grid[col + bottom_row_offset] + current_grid[right_col + bottom_row_offset] +
               current_grid[left_col + row_offset] + current_grid[right_col + row_offset];
    }

    __global__ void compute_next_gen_kernel(bool *current_grid, bool *next_grid, int N) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t row_offset = row * N;
        int index = row_offset + col;
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        int living_neighbors = kernels::count_neighbors(current_grid, col, row, N);
        next_grid[index] =
            living_neighbors == 3 || (living_neighbors == 2 && current_grid[index]) ? true : false;
        return;
    }

    __host__ __device__ int cyclic_check_neighbors(uint8_t *currentGrid, int col, int row, int grid_size, int index) {
        int leftCol = (col - 1 + grid_size) % grid_size;
        int rightCol = (col + 1) % grid_size;
        int rowOffset = row * grid_size;
        int topRowOffset = ((row - 1 + grid_size) % grid_size) * grid_size;
        int bottomRowOffset = ((row + 1) % grid_size) * grid_size;
        int nextState = (currentGrid[index] + 1) % TOTAL_STATES;

        return (
            ( currentGrid[col + topRowOffset] == nextState )
            || ( currentGrid[col + bottomRowOffset] == nextState )
            || ( currentGrid[rowOffset + leftCol] == nextState )
            || ( currentGrid[rowOffset + rightCol] == nextState )
        );
    }


    __global__ void cyclic_baseline_kernel(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t rowOffset = row * N;
        int index = rowOffset + col;
        int current_cell = currentGrid[index];
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        int nextStateNeighbor = kernels::cyclic_check_neighbors(currentGrid, col, row, N, index);
        if (current_cell == STATE1 && nextStateNeighbor) {
            nextGrid[index] = STATE2;
        } else if (current_cell == STATE2 && nextStateNeighbor) {
            nextGrid[index] = STATE3;
        } else if (current_cell == STATE3 && nextStateNeighbor) {
            nextGrid[index] = STATE4;
        } else if (current_cell == STATE4 && nextStateNeighbor) {
            nextGrid[index] = STATE5;
        } else if (current_cell == STATE5 && nextStateNeighbor) {
            nextGrid[index] = STATE6;
        } else if (current_cell == STATE6 && nextStateNeighbor) {
            nextGrid[index] = STATE7;
        } else if (current_cell == STATE7 && nextStateNeighbor) {
            nextGrid[index] = STATE8;
        } else if (current_cell == STATE8 && nextStateNeighbor) {
            nextGrid[index] = STATE9;
        } else if (current_cell == STATE9 && nextStateNeighbor) {
            nextGrid[index] = STATE10;
        } else if (current_cell == STATE10 && nextStateNeighbor) {
            nextGrid[index] = STATE11;
        } else if (current_cell == STATE11 && nextStateNeighbor) {
            nextGrid[index] = STATE12;
        } else if (current_cell == STATE12 && nextStateNeighbor) {
            nextGrid[index] = STATE13;
        } else if (current_cell == STATE13 && nextStateNeighbor) {
            nextGrid[index] = STATE14;
        } else if (current_cell == STATE14 && nextStateNeighbor) {
            nextGrid[index] = STATE15;
        } else if (current_cell == STATE15 && nextStateNeighbor) {
            nextGrid[index] = STATE1;
        } else {
            nextGrid[index] = current_cell;
        }
        return;
    }

    __global__ void cyclic_lookup_kernel(uint8_t *currentGrid, uint8_t* nextGrid, int N, uint8_t* lookup_table) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t rowOffset = row * N;
        int index = rowOffset + col;
        int current_cell = currentGrid[index];
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        uint8_t nextStateNeighbor = kernels::cyclic_check_neighbors(currentGrid, col, row, N, index);
        // printf("(%d, %d, %d)\n", row, col, lookup_table[current_cell * 2 + nextStateNeighbor]);
        nextGrid[index] = lookup_table[current_cell * 2 + nextStateNeighbor];
    }

    __device__ uint8_t getSubCellD(uint64_t currentCell, char position) {
        return (currentCell >> ((ELEMENTS_PER_CELL - 1 - position) * 8)) & 0xFF;
    }

    __device__ void setSubCellD(uint64_t *currentCell, char position, uint8_t subCell) {
        uint64_t mask = 0xFF;
        uint64_t newCellMask = subCell;
        
        // Erase pos content in cell:
        mask = mask << (ELEMENTS_PER_CELL - 1 - position) * 8;
        mask = ~mask;
        *currentCell = *currentCell & mask;
        
        // Add subcell content to cell in pos:
        *currentCell = *currentCell | (newCellMask << (ELEMENTS_PER_CELL - 1 - position) * 8);
    }

__global__ void cyclic_packet_coding_kernel(uint64_t *currentGrid, uint64_t* nextGrid, int N, uint8_t* lookup_table) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < GRID_SIZE && col < ROW_SIZE) {
        int leftCol = (col - 1 + (ROW_SIZE)) % ROW_SIZE;
        int rightCol = (col + 1) % ROW_SIZE;
        int rowOffset = row * ROW_SIZE;
        int topRowOffset = ((row - 1 + GRID_SIZE) % GRID_SIZE) * ROW_SIZE;
        int bottomRowOffset = ((row + 1) % GRID_SIZE) * ROW_SIZE;

        int index = row * ROW_SIZE + col;
        uint64_t currentCell = currentGrid[index];
        uint64_t nextCell = 0;

        // First subcell
        uint8_t subcell = getSubCellD(currentCell, 0);
        uint64_t upCell = currentGrid[col + topRowOffset];
        uint64_t downCell = currentGrid[col + bottomRowOffset];
        uint64_t leftCell = currentGrid[rowOffset + leftCol];
        uint64_t rightCell = currentGrid[rightCol + rowOffset];

        int nextStateNeighbor =
            (getSubCellD(upCell, 0) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(downCell, 0) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(leftCell, ELEMENTS_PER_CELL - 1) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(currentCell, 1) == (subcell + 1) % TOTAL_STATES);

        setSubCellD(&nextCell, 0, lookup_table[subcell * 2 + nextStateNeighbor]);

        for (int k = 1; k < ELEMENTS_PER_CELL - 1; k++) {
            subcell = getSubCellD(currentCell, k);
            nextStateNeighbor =
                (getSubCellD(upCell, k) == (subcell + 1) % TOTAL_STATES) ||
                (getSubCellD(downCell, k) == (subcell + 1) % TOTAL_STATES) ||
                (getSubCellD(currentCell, k - 1) == (subcell + 1) % TOTAL_STATES) ||
                (getSubCellD(currentCell, k + 1) == (subcell + 1) % TOTAL_STATES);

            setSubCellD(&nextCell, k, lookup_table[subcell * 2 + nextStateNeighbor]);
        }

        subcell = getSubCellD(currentCell, ELEMENTS_PER_CELL - 1);
        nextStateNeighbor =
            (getSubCellD(upCell, ELEMENTS_PER_CELL - 1) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(downCell, ELEMENTS_PER_CELL - 1) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(currentCell, ELEMENTS_PER_CELL - 2) == (subcell + 1) % TOTAL_STATES) ||
            (getSubCellD(rightCell, 0) == (subcell + 1) % TOTAL_STATES);

        setSubCellD(&nextCell, ELEMENTS_PER_CELL - 1, lookup_table[subcell * 2 + nextStateNeighbor]);
        nextGrid[index] = nextCell;
    }
}
} // namespace kernels

void compute_next_gen(bool *current_grid, bool *next_grid, size_t ca_grid_size) {
    // Allocate device memory
    bool *d_current = nullptr, *d_next = nullptr;
    size_t total_size = ca_grid_size * ca_grid_size;
    CUDA_CHECK(hipMalloc(&d_current, total_size * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_next, total_size * sizeof(bool)));

    // Copy data to device
    CUDA_CHECK(
        hipMemcpy(d_current, current_grid, total_size * sizeof(bool), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 block_size(32, 32);
    dim3 grid_size((ca_grid_size + block_size.x - 1) / block_size.x, (ca_grid_size + block_size.y - 1) / block_size.y);
    kernels::compute_next_gen_kernel<<<grid_size, block_size>>>(d_current, d_next, ca_grid_size);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(next_grid, d_next, total_size * sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
}

void cyclic_baseline(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
    // Allocate device memory
    uint8_t *d_current, *d_next;
    int totalSize = N * N;
    CUDA_CHECK(hipMalloc(&d_current, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_next, totalSize * sizeof(uint8_t)));

    // Copy data to device
    CUDA_CHECK(
        hipMemcpy(d_current, currentGrid, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    kernels::cyclic_baseline_kernel<<<gridSize, blockSize>>>(d_current, d_next, N);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(nextGrid, d_next, totalSize * sizeof(uint8_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
}

void cyclic_lookup_gen(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
    uint8_t *d_current, *d_next, lookup_table[TOTAL_STATES][2], *d_lookup_table;
    int totalSize = N * N;
    CUDA_CHECK(hipMalloc(&d_current, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_next, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2));

    CyclicCA::create_lookup_table(lookup_table);

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_current, currentGrid, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(d_lookup_table, lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    kernels::cyclic_lookup_kernel<<<gridSize, blockSize>>>(d_current, d_next, N, d_lookup_table);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(nextGrid, d_next, totalSize * sizeof(uint8_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
    CUDA_CHECK(hipFree(d_lookup_table));
}

uint8_t getSubCellH(uint64_t currentCell, char position) {
    return (currentCell >> (ELEMENTS_PER_CELL - 1 - position) * 8);
}

void setSubCellH(uint64_t* currentCell, char position, uint8_t subCell) {
    uint64_t mask = 0xFF;
    uint64_t newCellMask = subCell;
    mask = mask << (ELEMENTS_PER_CELL - 1 - position) * 8;
    mask = ~mask;
    *currentCell = *currentCell & mask;
    *currentCell = *currentCell | (newCellMask << (ELEMENTS_PER_CELL - 1 - position) * 8);
}

void cyclic_packet_coding_gen(uint64_t *currentGrid, uint64_t *nextGrid, int N) {
    uint64_t *d_current, *d_next;
    uint8_t lookup_table[TOTAL_STATES][2], *d_lookup_table;
    int totalSize = N * N;
    CUDA_CHECK(hipMalloc(&d_current, totalSize * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc(&d_next, totalSize * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc(&d_lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2));

    CyclicCA::create_lookup_table(lookup_table);

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_current, currentGrid, totalSize * sizeof(uint64_t), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(d_lookup_table, lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    kernels::cyclic_packet_coding_kernel<<<gridSize, blockSize>>>(d_current, d_next, N, d_lookup_table);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(nextGrid, d_next, totalSize * sizeof(uint64_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
    CUDA_CHECK(hipFree(d_lookup_table));
}
