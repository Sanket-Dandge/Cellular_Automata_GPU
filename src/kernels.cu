#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "kernels.hpp"
#include "cyclic_ca.hpp"
#include "utils.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                                           \
    {                                                                                              \
        hipError_t err = (call);                                                                  \
        if (err != hipSuccess) {                                                                  \
            printf("CUDA Error: %s at %s: %d\n", hipGetErrorString(err), __FILE__, __LINE__);     \
            exit(EXIT_FAILURE);                                                                    \
        }                                                                                          \
    }

namespace kernels {
    __host__ __device__ int count_neighbors(bool *current_grid, int col, int row, int grid_size) {
        int left_col = (col - 1 + grid_size) % grid_size;
        int right_col = (col + 1) % grid_size;
        int row_offset = row * grid_size;
        int top_row_offset = ((row - 1 + grid_size) % grid_size) * grid_size;
        int bottom_row_offset = ((row + 1) % grid_size) * grid_size;

        return current_grid[left_col + top_row_offset] + current_grid[col + top_row_offset] +
               current_grid[right_col + top_row_offset] + current_grid[left_col + bottom_row_offset] +
               current_grid[col + bottom_row_offset] + current_grid[right_col + bottom_row_offset] +
               current_grid[left_col + row_offset] + current_grid[right_col + row_offset];
    }

    __global__ void compute_next_gen_kernel(bool *current_grid, bool *next_grid, int N) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t row_offset = row * N;
        int index = row_offset + col;
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        int living_neighbors = kernels::count_neighbors(current_grid, col, row, N);
        next_grid[index] =
            living_neighbors == 3 || (living_neighbors == 2 && current_grid[index]) ? true : false;
        return;
    }

    __host__ __device__ int cyclic_check_neighbors(uint8_t *currentGrid, int col, int row, int grid_size, int index) {
        int leftCol = (col - 1 + grid_size) % grid_size;
        int rightCol = (col + 1) % grid_size;
        int rowOffset = row * grid_size;
        int topRowOffset = ((row - 1 + grid_size) % grid_size) * grid_size;
        int bottomRowOffset = ((row + 1) % grid_size) * grid_size;
        int nextState = (currentGrid[index] + 1) % TOTAL_STATES;

        return (
            ( currentGrid[col + topRowOffset] == nextState )
            || ( currentGrid[col + bottomRowOffset] == nextState )
            || ( currentGrid[rowOffset + leftCol] == nextState )
            || ( currentGrid[rowOffset + rightCol] == nextState )
        );
    }


    __global__ void cyclic_baseline_kernel(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t rowOffset = row * N;
        int index = rowOffset + col;
        int current_cell = currentGrid[index];
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        int nextStateNeighbor = kernels::cyclic_check_neighbors(currentGrid, col, row, N, index);
        if (current_cell == STATE1 && nextStateNeighbor) {
            nextGrid[index] = STATE2;
        } else if (current_cell == STATE2 && nextStateNeighbor) {
            nextGrid[index] = STATE3;
        } else if (current_cell == STATE3 && nextStateNeighbor) {
            nextGrid[index] = STATE4;
        } else if (current_cell == STATE4 && nextStateNeighbor) {
            nextGrid[index] = STATE5;
        } else if (current_cell == STATE5 && nextStateNeighbor) {
            nextGrid[index] = STATE6;
        } else if (current_cell == STATE6 && nextStateNeighbor) {
            nextGrid[index] = STATE7;
        } else if (current_cell == STATE7 && nextStateNeighbor) {
            nextGrid[index] = STATE8;
        } else if (current_cell == STATE8 && nextStateNeighbor) {
            nextGrid[index] = STATE9;
        } else if (current_cell == STATE9 && nextStateNeighbor) {
            nextGrid[index] = STATE10;
        } else if (current_cell == STATE10 && nextStateNeighbor) {
            nextGrid[index] = STATE11;
        } else if (current_cell == STATE11 && nextStateNeighbor) {
            nextGrid[index] = STATE12;
        } else if (current_cell == STATE12 && nextStateNeighbor) {
            nextGrid[index] = STATE13;
        } else if (current_cell == STATE13 && nextStateNeighbor) {
            nextGrid[index] = STATE14;
        } else if (current_cell == STATE14 && nextStateNeighbor) {
            nextGrid[index] = STATE15;
        } else if (current_cell == STATE15 && nextStateNeighbor) {
            nextGrid[index] = STATE1;
        } else {
            nextGrid[index] = current_cell;
        }
        return;
    }

    // TODO: Complete It
    __global__ void cyclic_lookup_kernel(uint8_t *currentGrid, uint8_t* nextGrid, int N, uint8_t* lookup_table) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        size_t rowOffset = row * N;
        int index = rowOffset + col;
        int current_cell = currentGrid[index];
        if (index >= N * N) {
            printf("%d,%d\n", col, row);
        }
        uint8_t nextStateNeighbor = kernels::cyclic_check_neighbors(currentGrid, col, row, N, index);
        // printf("(%d, %d, %d)\n", row, col, lookup_table[current_cell * 2 + nextStateNeighbor]);
        nextGrid[index] = lookup_table[current_cell * 2 + nextStateNeighbor];
    }
} // namespace kernels

void compute_next_gen(bool *current_grid, bool *next_grid, size_t ca_grid_size) {
    // Allocate device memory
    bool *d_current = nullptr, *d_next = nullptr;
    size_t total_size = ca_grid_size * ca_grid_size;
    CUDA_CHECK(hipMalloc(&d_current, total_size * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_next, total_size * sizeof(bool)));

    // Copy data to device
    CUDA_CHECK(
        hipMemcpy(d_current, current_grid, total_size * sizeof(bool), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 block_size(32, 32);
    dim3 grid_size((ca_grid_size + block_size.x - 1) / block_size.x, (ca_grid_size + block_size.y - 1) / block_size.y);
    kernels::compute_next_gen_kernel<<<grid_size, block_size>>>(d_current, d_next, ca_grid_size);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(next_grid, d_next, total_size * sizeof(bool), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
}

void cyclic_baseline(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
    // Allocate device memory
    uint8_t *d_current, *d_next;
    int totalSize = N * N;
    CUDA_CHECK(hipMalloc(&d_current, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_next, totalSize * sizeof(uint8_t)));

    // Copy data to device
    CUDA_CHECK(
        hipMemcpy(d_current, currentGrid, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice));

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    kernels::cyclic_baseline_kernel<<<gridSize, blockSize>>>(d_current, d_next, N);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(nextGrid, d_next, totalSize * sizeof(uint8_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
}

void cyclic_lookup_gen(uint8_t *currentGrid, uint8_t *nextGrid, int N) {
    uint8_t *d_current, *d_next, lookup_table[TOTAL_STATES][2], *d_lookup_table;
    int totalSize = N * N;
    CUDA_CHECK(hipMalloc(&d_current, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_next, totalSize * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc(&d_lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2));

    CyclicCA::create_lookup_table(lookup_table);

    // Copy data to device
    CUDA_CHECK(
        hipMemcpy(d_current, currentGrid, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(d_lookup_table, lookup_table, TOTAL_STATES * sizeof(uint8_t) * 2, hipMemcpyHostToDevice));

    // Launch kernel
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    // int linGrid = (int)ceil(GRID_SIZE/(float)32);
    // dim3 gridSize(linGrid,linGrid);
    kernels::cyclic_lookup_kernel<<<gridSize, blockSize>>>(d_current, d_next, N, d_lookup_table);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(nextGrid, d_next, totalSize * sizeof(uint8_t), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_current));
    CUDA_CHECK(hipFree(d_next));
    CUDA_CHECK(hipFree(d_lookup_table));
}
